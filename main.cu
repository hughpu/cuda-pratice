#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void mykernel() {
    printf("Hello, World!\n");
}

int main()
{
    mykernel<<<1, 1>>>();
    hipDeviceSynchronize();

    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
      // print the CUDA error message and exit
      printf("CUDA error: %s\n", hipGetErrorString(error));
      exit(-1);
    }
    return 0;
}