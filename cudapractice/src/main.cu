#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/universal_vector.h>

#include <cstdio>
#include <cudapractice/helper_cuda.cuh>
#include <map>
#include <string>
#include <vector>

#define USAGE                                                                \
  "USAGE:\n    ./main <demo_name>\n    demo_name: hello_world, reduce_sum, " \
  "transpose, gemm"

template <class T>
class CudaAllocator {
 public:
  using value_type = T;

  T *allocate(size_t size) {
    T *ptr = nullptr;
    checkCudaErrors(hipMallocManaged(&ptr, size * sizeof(T)));
    return ptr;
  }

  void deallocate(T *ptr, size_t size) { checkCudaErrors(hipFree(ptr)); }

  template <class... Args>
  void construct(T *ptr, Args &&...args) {
    if constexpr (!(sizeof...(Args) == 0 && std::is_pod_v<T>)) {
      ::new (ptr) T(std::forward<Args>(args)...);
    }
  }
};

template <class Func>
__global__ void MyKernel(int n, Func func) {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x; i < n;
       i += blockDim.x * gridDim.x) {
    func(i);
  }
}

template <int blockSize, class T>
__global__ void ReduceSumKernel(int n, T *sum, T *arr) {
  __shared__ volatile T local_sum[blockSize];
  int i = blockIdx.x;
  int j = threadIdx.x;
  T temp_sum = 0;
  for (int t = j + blockSize * i; t < n; t += blockSize * gridDim.x) {
    temp_sum += arr[t];
  }

  local_sum[j] = temp_sum;
  __syncthreads();

  if constexpr (blockSize >= 1024) {
    if (j < 512) {
      local_sum[j] += local_sum[j + 512];
    }
    __syncthreads();
  }
  if constexpr (blockSize >= 512) {
    if (j < 256) {
      local_sum[j] += local_sum[j + 256];
    }
    __syncthreads();
  }
  if constexpr (blockSize >= 256) {
    if (j < 128) {
      local_sum[j] += local_sum[j + 128];
    }
  }
  if constexpr (blockSize >= 128) {
    if (j < 64) {
      local_sum[j] += local_sum[j + 64];
    }
    __syncthreads();
  }
  if (j < 32) {
    if constexpr (blockSize >= 64) {
      local_sum[j] += local_sum[j + 32];
    }
    if constexpr (blockSize >= 32) {
      local_sum[j] += local_sum[j + 16];
    }
    if constexpr (blockSize >= 16) {
      local_sum[j] += local_sum[j + 8];
    }
    if constexpr (blockSize >= 8) {
      local_sum[j] += local_sum[j + 4];
    }
    if constexpr (blockSize >= 4) {
      local_sum[j] += local_sum[j + 2];
    }
    if (j == 0) {
      sum[i] = local_sum[0] + local_sum[1];
    }
  }
}

void HelloWorld() {
  int n = 65536;
  std::vector<float, CudaAllocator<float>> y(n);
  thrust::host_vector<float> x_host(n);

  float a = 3.14f;

  for (int i = 0; i < n; ++i) {
    y[i] = std::rand() * (1.f / RAND_MAX);
  }

  thrust::generate(x_host.begin(), x_host.end(),
                   []() { return std::rand() * (1.f / RAND_MAX); });

  thrust::device_vector<float> x_dev = x_host;
  MyKernel<<<32, 128>>>(n, [a, x = x_dev.data(), y = y.data()] __device__(
                               int i) { x[i] = a * x[i] + y[i] + __sinf(i); });

  // checkCudaErrors(hipDeviceSynchronize());
  x_host = x_dev;

  for (int i = 0; i < n; i++) {
    printf("arr[%d]: %f\n", i, x_host[i]);
  }
}

template <int reduceScale = 4096, int blockSize = 256, class T>
T ReduceSum(thrust::universal_vector<T> &arr, int n) {
  thrust::universal_vector<T> x(n);
  thrust::universal_vector<T> sum(n / reduceScale);
  ReduceSumKernel<blockSize>
      <<<n / reduceScale, blockSize>>>(n, sum.data().get(), arr.data().get());
  checkCudaErrors(hipDeviceSynchronize());
  T final_sum = 0;
  for (int i = 0; i < n / reduceScale; ++i) {
    final_sum += sum[i];
  }
  return final_sum;
}

void ReduceSum() {
  int n = 1 << 24;
  thrust::universal_vector<int> arr(n);
  for (int i = 0; i < n; ++i) {
    arr[i] = std::rand() % 4;
  }

  float final_sum = ReduceSum(arr, n);
  printf("final sum: %f\n", final_sum);
}

enum Demo {
  REDUCE_SUM,
  HELLO_WORLD,
  GEMM,
  TRANSPOSE,
};

int main(int argc, char *argv[]) {
  std::map<std::string, Demo> demo_map = {{
                                              "reduce_sum",
                                              REDUCE_SUM,
                                          },
                                          {

                                              "hello_world",
                                              HELLO_WORLD,
                                          },
                                          {
                                              "gemm",
                                              GEMM,
                                          },
                                          {
                                              "transpose",
                                              TRANSPOSE,
                                          }};
  if (argc != 2) {
    printf(USAGE);
    return 1;
  }

  if (demo_map.find(std::string(argv[1])) == demo_map.end()) {
    printf(USAGE);
    return 1;
  }

  Demo demo_name = demo_map[std::string(argv[1])];

  switch (demo_name) {
    case Demo::REDUCE_SUM:
      ReduceSum();
      break;
    case Demo::HELLO_WORLD:
      HelloWorld();
      break;
    case Demo::TRANSPOSE:
      printf("transpose is not implemented yet");
      break;
    case Demo::GEMM:
      printf("gemm is not implemented yet");
      break;
    default:
      printf(USAGE);
      return 1;
  }

  return 0;
}